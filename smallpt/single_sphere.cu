#include "hip/hip_runtime.h"
// based on smallpt, a path tracer by Kevin Beason, 2008  
 
#include <iostream>
#include <chrono>  // for high_resolution_clock
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include ""
#include "helper_string.h"
#include "ray.h"


#define CUDA_CALL_CHECK(x)                             \
do{                                                    \
    hipError_t cudaStatus = x;                        \
    if((cudaStatus) != hipSuccess)                    \
    {                                                  \
        printf("Error at %s:%d\t",__FILE__,__LINE__);  \
        printf("%s %d\t",#x, (cudaStatus));            \
        printf("%s\n",hipGetErrorString(cudaStatus)); \
        system("pause");                               \
        return EXIT_FAILURE;                           \
    }                                                  \
} while(0)


#define M_PI 3.14159265359f  // pi


__device__ float hit_sphere(const ray& r, const vec3& center, float radius)
{
    vec3 oc = r.origin() - center;
    float a = dot(r.direction(), r.direction());
    float b = 2.0f * dot(oc, r.direction());
    float c = dot(oc, oc) - radius * radius;
    float d = b * b - 4.f * a * c;
    if (d < 0.f)
    {
        return -1.f;
    }
    else
    {
        return (-b - sqrt(d)) / (2.f * a);
    }
}

__device__ vec3 color(const ray& r)
{
    float t = hit_sphere(r, vec3(0.f, 0.f, -1.f), 0.5f);
    if (t > 0.f)
    {
        vec3 n = unit_vector(r.point_at_parameter(t) - vec3(0.f, 0.f, -1.f));
        return 0.5f * vec3(n.x() + 1.f, n.y() + 1.f, n.z() + 1.f);
    }
    else
    {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5*(unit_direction.y() + 1.0);
        return (1.0 - t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
    }
}

#define SUB_GRID_X 256
#define SUB_GRID_Y 256                
#define BLOCK_X 8
#define BLOCK_Y 8

__global__ void setup_random_kernel(hiprandState *states, int nx, int ny, int subx = 0, int suby = 0)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    x = SUB_GRID_X * subx + x;
    y = SUB_GRID_Y * suby + y;
    unsigned int i = (ny - y - 1) * nx + x; // index of current pixel (calculated using thread index) 

    hiprand_init(1234, i, 0, &states[i]);
}

// __global__ : executed on the device (GPU) and callable only from host (CPU) 
__global__ void render_kernel(hiprandState *states, float* output, int nx, int ny, int ns, int subx = 0, int suby = 0)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    x = SUB_GRID_X * subx + x;
    y = SUB_GRID_Y * suby + y;
    unsigned int i = (ny - y - 1) * nx + x; // index of current pixel (calculated using thread index) 

    vec3 low_left_corner(-1.f, -1.f, -1.f);
    vec3 horizonal(2.f, 0.f, 0.f);
    vec3 vertical(0.f, 2.f, 0.f);
    vec3 origin(0.f, 0.f, 0.f);

    hiprandState localState = states[i];
    vec3 *pic = (vec3*)output;
    vec3 col(0.f, 0.f, 0.f);
    for (int s = 0; s < ns; s++)
    {
        float dx = hiprand_uniform(&localState);
        float dy = hiprand_uniform(&localState);
        float u = float(x + dx) / float(nx);
        float v = float(y + dy) / float(ny);
        ray r(origin, low_left_corner + u * horizonal + v * vertical);
        col += color(r);
    }
    states[i] = localState;
    col /= float(ns);
    pic[i] = col;
}

inline float clamp(float x){ return x < 0.0f ? 0.0f : x > 1.0f ? 1.0f : x; } 

inline int toInt(float x){ return int(pow(clamp(x), 1 / 2.2) * 255 + .5); }  // convert RGB float in range [0,1] to int in range [0, 255] and perform gamma correction

void SaveToPPM(float* output, int w, int h);

void save_curandState(void* output, int w, int h)
{
    char filename[128];
    sprintf_s(filename, 128, "curand_state_%dx%d.bin", w, h);
    FILE *f = fopen(filename, "wb");          
    fwrite(output,sizeof(hiprandState), w * h, f);
    fclose(f);
    fprintf(stdout, "- save hiprand states into %s\n", filename);
}

void load_curandState(void* output, int w, int h)
{
    char filename[128];
    sprintf_s(filename, 128, "curand_state_%dx%d.bin", w, h);
    FILE *f = fopen(filename, "wb");          
    fread(output,sizeof(hiprandState), w * h, f);
    fclose(f);
    fprintf(stdout, "- load hiprand states from %s\n", filename);
}

inline bool file_exists(const std::string name)
{
    struct stat buffer;
    return (stat(name.c_str(), &buffer) == 0);
}

int TestSmallPTOnGPU(int width, int height, int samps)
{
    printf("\n Param  %d, %d, %d\n", width, height, samps);

    float* output_h = new float[width * height * 3]; // pointer to memory for image on the host (system RAM)
    float* output_d;    // pointer to memory for image on the device (GPU VRAM)

    std::chrono::duration<double> elapsed;

    int nSubx = (width + SUB_GRID_X - 1) / SUB_GRID_X;
    int nSuby = (width + SUB_GRID_Y - 1) / SUB_GRID_Y;

    dim3 block(BLOCK_X, BLOCK_Y, 1);   
    dim3 subgrid(SUB_GRID_X / block.x, SUB_GRID_Y / block.y, 1);
    dim3 grid( width/ block.x, height / block.y, 1);


    CUDA_CALL_CHECK( hipMalloc(&output_d, width * height * sizeof(float) * 3) );
        
    hiprandState *devStates;
    hiprandState *devStates_h = new hiprandState[width * height ];
    CUDA_CALL_CHECK(hipMalloc((void **)&devStates, width * height * sizeof(hiprandState)));

    char curandState_filename[128];
    sprintf_s(curandState_filename, 128, "curand_state_%dx%d.bin", width, height);
    if (!file_exists(curandState_filename))
    {
        // Record start time                          
        auto startRand = std::chrono::high_resolution_clock::now();

        for (int i = 0; i < nSuby; ++i)
        {
            fprintf(stdout, "\rGenerate hiprand state(%5.2lf%%)...\n", 100.f * float(i) / float(nSuby));
            for (int j = 0; j < nSubx; ++j)
            {
                setup_random_kernel <<< subgrid, block >>>(devStates, width, height, j, i); 
                CUDA_CALL_CHECK(hipGetLastError());
            }
        }
                CUDA_CALL_CHECK(hipDeviceSynchronize());
        auto finishRand = std::chrono::high_resolution_clock::now();
        elapsed = finishRand - startRand;
        printf("Random State Done! Time=%lf seconds\n", elapsed.count());

        CUDA_CALL_CHECK(hipMemcpy(devStates_h , devStates, width * height * sizeof(hiprandState), hipMemcpyDeviceToHost));
        save_curandState(devStates_h, width, height);
    }
    else
    {
        load_curandState(devStates_h, width, height);
        CUDA_CALL_CHECK(hipMemcpy(devStates, devStates_h , width * height * sizeof(hiprandState), hipMemcpyHostToDevice));
    }


    printf("\nStart rendering... %d, %d, %d\n", width, height, samps);
 
    // Record start time                          
    auto start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < nSubx; ++i)
        for (int j = 0; j < nSuby; ++j)
            render_kernel <<< grid, block >>>(devStates, output_d, width, height, samps);  
    CUDA_CALL_CHECK(hipGetLastError());
    CUDA_CALL_CHECK(hipDeviceSynchronize());

    // Record end time
    auto finish = std::chrono::high_resolution_clock::now();
    elapsed = finish - start;
    printf("Render Done! Time=%lf seconds\n", elapsed.count());

    CUDA_CALL_CHECK(hipMemcpy(output_h, output_d, width * height * sizeof(float3), hipMemcpyDeviceToHost));
    CUDA_CALL_CHECK( hipFree(output_d) );  

    SaveToPPM(output_h, width, height);

    printf("Saved image to 'smallptcuda.ppm'\n");
    delete[] output_h;
    return 0;
}

int main(int argc, char *argv[])
{
    ///int width = 512, height = 512, samps = 1024;
    int width = 512, height = 512, samps = 128;
    
    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **)argv, "width"))
            width = getCmdLineArgumentInt(argc, (const char **)argv, "width");
        if (checkCmdLineFlag(argc, (const char **)argv, "height"))
            height = getCmdLineArgumentInt(argc, (const char **)argv, "height");
        if (checkCmdLineFlag(argc, (const char **)argv, "samples"))
            samps = getCmdLineArgumentInt(argc, (const char **)argv, "samples");
    }
    CUDA_CALL_CHECK( hipSetDevice(0) );

    TestSmallPTOnGPU(width, height, samps);
    system("PAUSE");
}

void SaveToPPM(float* output, int w, int h)
{
    // Write image to PPM file, a very simple image file format
    FILE *f = fopen("smallptcuda.ppm", "w");          
    fprintf(f, "P3\n%d %d\n%d\n", w, h, 255);
    for (int i = 0; i < w * h * 3; i += 3)  // loop over pixels, write RGB values
    fprintf(f, "%d %d %d ", toInt(output[i]), toInt(output[i + 1]), toInt(output[i + 2]));
    fclose(f);

    system("ffplay smallptcuda.ppm");
}